#include "hip/hip_runtime.h"
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = a * x[i] + y[i];
}

int main(void) {
  int N = 1 << 3;
  float *x, *y, *d_x, *d_y;
  hipError_t err;

  x = (float *) malloc(N * sizeof(float));
  y = (float *) malloc(N * sizeof(float));

  err = hipMalloc(&d_x, N * sizeof(float));
  gpuErrchk(err);
  err = hipMalloc(&d_y, N * sizeof(float));
  gpuErrchk(err);

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  err = hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  gpuErrchk(err);
  err = hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
  gpuErrchk(err);

  saxpy<<<(N + 255) / 256, 25600>>>(N, 2.0f, d_x, d_y);
  gpuErrchk( hipPeekAtLastError() );

  err = hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  gpuErrchk(err);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i] - 4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}